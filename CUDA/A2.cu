
/*  Assignment 2 
	Block Wise reduction
	Author: Parth Tiwari
	Roll: 16IM30025
	Date: 26th Feb 2020
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h> 
#include <hip/hip_runtime.h>


__global__ void reduce(float* A, float* B, int q)
{
 	
	int num_threads = blockDim.x;
	int block_num = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = block_num * num_threads + threadIdx.x;

 	for(int stride = 1; stride < num_threads; stride *= 2)
 	{
 		__syncthreads();

 		if(tid % (2* stride) == 0)
 		{
 			A[tid] += A[tid + stride];
 		}
 	}

 	if(tid % q == 0)
 	{
 		B[block_num] = A[tid]/q;
 	}

}


int main()
{
	int T, p, q;

	scanf("%d", &T);

	while(T--)
	{
		scanf("%d %d", &p, &q);

		p = pow(2, p);
		q = pow(2, q);

		size_t size = p*sizeof(float);

		float *A;
		float *B;
		A = (float*)malloc(size);
		B = (float*)malloc(size/q);

		for(int i = 0; i < p; i++)
		{
			scanf("%f", &A[i]);
		}

		hipError_t err = hipSuccess;

		// Declare array for device
		float* A_d = NULL;
		float* B_d = NULL;

		err = hipMalloc((void**)&A_d, size);
		err = hipMalloc((void**)&B_d, size/q);

		//copy memory to device
	  	err = hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
  		err = hipMemcpy(B_d, B, size/q, hipMemcpyHostToDevice);


	  	int flag = 1;
	  	while(p >= q)
	  	{
			dim3 grid(ceil(sqrt(p/q)),ceil(sqrt(p/q)),1);
			dim3 block(q,1,1);

			if(flag == 1)
				reduce<<<grid,block>>>(A_d, B_d, q);
			else
				reduce<<<grid,block>>>(B_d, A_d, q);

			p /= q;
			size /= q;
			flag *= -1;
		}

		if(flag == -1)
		{
			err = hipMemcpy(B, B_d, size, hipMemcpyDeviceToHost);
			for(int i = 0; i < p; i++)
				printf("%.2f ", B[i]);
		}
		else
		{
			err = hipMemcpy(A, A_d, size, hipMemcpyDeviceToHost);
			for(int i = 0; i < p; i++)
				printf("%.2f ", A[i]);	
		}

		hipFree(A_d);
		hipFree(B_d);
		free(A);
		free(B);

		printf("\n");
	}
	return 0;
}